#include "hip/hip_runtime.h"
// Compile: nvcc Hello_world.cu -o Hello_world
// Run:     ./Hello_world

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void hello_world(int *global_m, int threadPerBlock) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    global_m[bid * threadPerBlock + tid] = 1;
}

int main() {

    // Test if CUDA available
    int deviceCount = 0, blockCount = 0, maxThreadsPerBlock = 0;

    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("There is no device suppporting CUDA\n");
    }
    int dev = 0;
    for (; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                printf("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
                printf("There is 1 device supporting CUDA.\n");
            else
                printf("there are %d devices supporting CUDA.\n", deviceCount);
        }
        printf("\nDevice %d:\"%s\"\n", dev, deviceProp.name);
        // printf("Major revision number: %d\n", deviceProp.major);
        // printf("Minor revision number: %d\n", deviceProp.minor);
        printf("Compute capability(version): %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("Total amount of global memory: %lu bytes (%.3f gigabytes)\n", deviceProp.totalGlobalMem, ((float)deviceProp.totalGlobalMem) * 1e-9);

#if CUDART_VERSION >= 2000
        printf("Number of multiprocessors(SM): %d\n", deviceProp.multiProcessorCount);
        printf("Number of cores: %d\n", 8 * deviceProp.multiProcessorCount);
#endif
        printf("Total amount of constant memory: %lu bytes\n", deviceProp.totalConstMem);
        printf("Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
        printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
        printf("Warp size(in threads): %d\n", deviceProp.warpSize);
        printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("Maximum sizes of each dimension of a block: %d x %d x %d \n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[1]);
        printf("Maximum sizes of each dimension of a grid: %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("Maximum memory pitch: %lu bytes\n", deviceProp.memPitch);
        printf("Texture alignment: %lu bytes\n", deviceProp.textureAlignment);
        printf("Clock rate: %.0f MHz\n", deviceProp.clockRate * 1e-3);
#if CUDART_VERSION >= 2000
        printf("Concurrent copy and execution: %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
#endif
        blockCount += deviceProp.multiProcessorCount;
        maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    }

    int *host_m, *global_m, sum = 0;

    // allocate host memory
    host_m = (int *)malloc(sizeof(int) * blockCount * maxThreadsPerBlock);

    // Allocate device memory
    hipMalloc((void **)&global_m, sizeof(int) * blockCount * maxThreadsPerBlock);

    // Execute kernels
    hello_world<<<blockCount, maxThreadsPerBlock>>>(global_m, maxThreadsPerBlock);

    // Transfer output from device memory to host
    hipMemcpy(host_m, global_m, sizeof(int) * blockCount * maxThreadsPerBlock, hipMemcpyDeviceToHost);

    for (int i = 0; i < blockCount * maxThreadsPerBlock; i++) {
        sum += host_m[i];
    }

    printf("\nHello_world!\n  ——  from %d CUDA threads in %d GPU(s)\n", sum, deviceCount);

    hipFree(global_m);
    free(host_m);

    return 0;
}
